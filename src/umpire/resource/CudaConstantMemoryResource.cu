//////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2016-23, Lawrence Livermore National Security, LLC and Umpire
// project contributors. See the COPYRIGHT file for details.
//
// SPDX-License-Identifier: (MIT)
//////////////////////////////////////////////////////////////////////////////
#include "umpire/resource/CudaConstantMemoryResource.hpp"

#include "umpire/ResourceManager.hpp"
#include "umpire/util/Macros.hpp"
#include "umpire/util/error.hpp"

#include <memory>
#include <sstream>

__constant__ static char s_umpire_internal_device_constant_memory[64*1024];

namespace umpire {
namespace resource {

CudaConstantMemoryResource::CudaConstantMemoryResource(const std::string& name, int id, MemoryResourceTraits traits) :
  MemoryResource{name, id, traits},
  m_current_size{0},
  m_highwatermark{0},
  m_platform{Platform::cuda},
  m_offset{0},
  m_ptr{nullptr},
  m_initialized{false}
{
}

void* CudaConstantMemoryResource::allocate(std::size_t bytes)
{
  std::lock_guard<std::mutex> lock{m_mutex};

  if (!m_initialized) {
    hipError_t error = ::hipGetSymbolAddress((void**)&m_ptr, s_umpire_internal_device_constant_memory);

    if (error != hipSuccess) {
      UMPIRE_ERROR(runtime_error, umpire::fmt::format("hipGetSymbolAddress failed with error: {}", hipGetErrorString(error)));
    }

    m_initialized = true;
  }

  char* ptr{static_cast<char*>(m_ptr) + m_offset};
  m_offset += bytes;

  void* ret{static_cast<void*>(ptr)};

  if (m_offset > (1024 * 64))
  {
    UMPIRE_ERROR(runtime_error, umpire::fmt::format("Max total size of constant allocations is 64KB, current size is {} bytes", (m_offset - bytes)));
  }

  UMPIRE_LOG(Debug, "(bytes=" << bytes << ") returning " << ret);

  return ret;
}

void CudaConstantMemoryResource::deallocate(void* ptr, std::size_t size)
{
  std::lock_guard<std::mutex> lock{m_mutex};

  UMPIRE_LOG(Debug, "(ptr=" << ptr << ")");

  if ( (static_cast<char*>(m_ptr) + (m_offset - size))
      == static_cast<char*>(ptr)) {
    m_offset -= size;
  } else {
    UMPIRE_ERROR(runtime_error,"CudaConstantMemory deallocations must be in reverse order");
  }
}

bool CudaConstantMemoryResource::isAccessibleFrom(Platform p) noexcept
{
  if(p == Platform::cuda)
    return true;
  else
    return false;
}

Platform CudaConstantMemoryResource::getPlatform() noexcept
{
  return m_platform;
}

} // end of namespace resource
} // end of namespace umpire
